#include "hip/hip_runtime.h"
// -*- mode:c++;indent-tabs-mode:nil;c-basic-offset:4;coding:utf-8 -*-
// vi: set et ft=c++ ts=4 sts=4 sw=4 fenc=utf-8 :vi
//
// Copyright 2024 Mozilla Foundation
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "tinyblas.h"

//
//                   _   _          ___ _      _   ___
//                  | |_(_)_ _ _  _| _ ) |    /_\ / __|
//                  |  _| | ' \ || | _ \ |__ / _ \\__ \.
//                   \__|_|_||_\_, |___/____/_/ \_\___/
//                             |__/
//
//                    BASIC LINEAR ALGEBRA SUBPROGRAMS
//
//
// In this file you'll find GPU subroutines implementing general matrix
// multiplication, that are API compatible with NVIDIA's cuBLAS library
// and nearly as fast[1] too. This is important because how can we call
// our software open source if it spends most of its time inside NVIDIA
// proprietary blobs like cuBLAS? tinyBLAS provides a free, open, libre
// alternative to cuBLAS that's orders of a magnitude tinier (cuBLAS is
// a 500mb DSO) and goes fast enough that you won't sacrifice much. AMD
// users might even prefer tinyBLAS, since outperforming hipBLAS is not
// very difficult for large matrices. tinyBLAS also has better accuracy
// too, since hipBLAS uses tricks that cause sign flips, and denormals.
//
// TODO(jart): make tinyBLAS go fast for skinny matrices
//
// [1] S. Boehm, ‘How to Optimize a CUDA Matmul Kernel for cuBLAS-like
//     Performance’, 2022. [Online]. Available:
//     https://siboehm.com/articles/22/CUDA-MMM. [Accessed:
//     05-Mar-2024].

#include <algorithm>
#include <cstdlib>
#include <type_traits>

#ifndef __HIP__
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#else
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#define hipSuccess hipSuccess
#define hipStream_t hipStream_t
#define hipGetLastError hipGetLastError
#endif

#define WARPSIZE 32
#define THREAD_COUNT ((BM * BN) / (TM * TN))
#define KERNEL __launch_bounds__(THREAD_COUNT)
#define CEIL_DIV(M, N) (((M) + (N) - 1) / (N))

////////////////////////////////////////////////////////////////////////////////////////////////////
// tinyBLAS block tiling outer product GEMM kernel

template <int BM, int BN, int TM, int TN, typename WORD, typename SRC, typename DST>
static __device__ void matmul_block2d(tinyblasOperation_t transa, tinyblasOperation_t transb, int m,
                                      int n, int k, WORD alpha, const SRC *A, int lda, const SRC *B,
                                      int ldb, WORD beta, DST *C, int ldc) {

    constexpr int BK = THREAD_COUNT;
    static_assert(BM % TM == 0, "can't divide work for threads");
    static_assert(BN % TN == 0, "can't divide work for threads");
    static_assert(BM > 0 && BN > 0 && BK > 0 && TM > 0 && TN > 0,
                  "one of the constexpr configuration values was non-positive");
    static_assert((BK * BM * sizeof(SRC)) + (BK * BN * sizeof(SRC)) <= 65536,
                  "you're almost almost certainly using too much shared memory");

    const int th = threadIdx.x;
    const int ii = blockIdx.x * BM;
    const int jj = blockIdx.y * BN;
    const int ti = th / (BN / TN) * TM;
    const int tj = th % (BN / TN) * TN;

    __shared__ SRC As[BK * BM];
    __shared__ SRC Bs[BK * BN];

    WORD At[TM];
    WORD Bt[TN];
    WORD Cs[TM * TN] = {0};

    for (int ll = 0; ll < k; ll += BK) {

        for (int i = 0; i < BM; ++i)
            As[BM * th + i] = 0;
        for (int i = 0; i < BM && ll + th < k && ii + i < m; ++i)
            As[BM * th + i] = A[transa ? lda * (ii + i) + (ll + th) : lda * (ll + th) + (ii + i)];

        for (int j = 0; j < BN; ++j)
            Bs[BN * th + j] = 0;
        for (int j = 0; j < BN && ll + th < k && jj + j < n; ++j)
            Bs[BN * th + j] = B[transb ? ldb * (ll + th) + (jj + j) : ldb * (jj + j) + (ll + th)];

        __syncthreads();

        for (int l = 0; l < BK; ++l) {
            for (int j = 0; j < TM; ++j)
                At[j] = As[BM * l + ti + j];
            for (int h = 0; h < TN; ++h)
                Bt[h] = Bs[BN * l + tj + h];
            for (int j = 0; j < TM; ++j) {
                WORD a = At[j];
                for (int h = 0; h < TN; ++h) {
                    WORD b = Bt[h];
                    Cs[TN * j + h] += a * b;
                }
            }
        }

        __syncthreads();
    }

    if (alpha != (WORD)1)
        for (int i = 0; i < TM * TN; ++i)
            Cs[i] *= alpha;

    for (int j = 0; j < TN && jj + tj + j < n; ++j)
        for (int i = 0; i < TM && ii + ti + i < m; ++i)
            if (beta) {
                WORD c = C[ldc * (jj + tj + j) + (ii + ti + i)];
                C[ldc * (jj + tj + j) + (ii + ti + i)] = c * beta + Cs[TN * i + j];
            } else {
                C[ldc * (jj + tj + j) + (ii + ti + i)] = Cs[TN * i + j];
            }
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// tinyBLAS warp block tiling outer product GEMM kernel

enum Mode {
    GENERAL,
    SIMPLE,
};

template <enum Mode MODE, int BM, int BN, int BK, int WM, int WN, int WNI, int TM, int TN, int TT,
          typename WORD, typename SRC, typename DST>
static __device__ void matmul_warp2d(tinyblasOperation_t aT, //
                                     tinyblasOperation_t bT, //
                                     int m, int n, int k, WORD alpha, //
                                     const SRC *A, int lda, //
                                     const SRC *B, int ldb, WORD beta, //
                                     DST *C, int ldc) {

    const int warpIdx = threadIdx.x / WARPSIZE;
    const int warpCol = warpIdx % (BN / WN);
    const int warpRow = warpIdx / (BN / WN);

    constexpr int WARPS = TT / WARPSIZE;
    constexpr int WMI = (WM * WN) / (WARPSIZE * TM * TN * WNI);
    constexpr int WSUBM = WM / WMI;
    constexpr int WSUBN = WN / WNI;
    constexpr int VE = sizeof(float4) / sizeof(SRC);

    const int threadIdxInWarp = threadIdx.x % WARPSIZE;
    const int threadColInWarp = threadIdxInWarp % (WSUBN / TN);
    const int threadRowInWarp = threadIdxInWarp / (WSUBN / TN);

    // want to tune these magnums?
    // use llamafile/pick_a_warp_kernel.c
    static_assert(!(BN % WN) && !(BM % WM), "");
    static_assert((BN / WN) * (BM / WM) == WARPS, "");
    static_assert(!((WM * WN) % (WARPSIZE * TM * TN * WNI)), "");
    static_assert(BN % (sizeof(float4) * TN) == 0, "");
    static_assert(BM % (sizeof(float4) * TM) == 0, "");
    static_assert(!(WM % WMI) && !(WN % WNI), "");
    static_assert(!((BM * BK) % (VE * TT)), "");
    static_assert(!((BN * BK) % (VE * TT)), "");

    __shared__ SRC As[BK * BM];
    __shared__ SRC Bs[BK * BN];

    WORD Ar[WMI * TM] = {0};
    WORD Br[WNI * TN] = {0};
    WORD Cr[WMI * TM * WNI * TN] = {0};

    for (int bkIdx = 0; bkIdx < k; bkIdx += BK) {

        for (int h = 0; h < BM; h += (TT * VE) / BK)
            for (int v = 0; v < VE; ++v) {
                int l = bkIdx + threadIdx.x % (BK / VE) * VE + v;
                int i = blockIdx.y * BM + threadIdx.x / (BK / VE) + h;
                As[BM * (threadIdx.x % (BK / VE) * VE + v) + (threadIdx.x / (BK / VE) + h)] =
                    aT && MODE <= GENERAL
                        ? ((MODE == SIMPLE ? i < m : (l < k && i < m)) ? A[lda * l + i] : (SRC)0)
                        : ((MODE == SIMPLE ? i < m : (l < k && i < m)) ? A[lda * i + l] : (SRC)0);
            }

        for (int h = 0; h < BK; h += TT / (BN / VE))
            for (int v = 0; v < VE; ++v) {
                int l = bkIdx + threadIdx.x / (BN / VE) + h;
                int j = blockIdx.x * BN + threadIdx.x % (BN / VE) * VE + v;
                Bs[BN * (threadIdx.x / (BN / VE) + h) + (threadIdx.x % (BN / VE) * VE + v)] =
                    bT || MODE >= SIMPLE
                        ? (MODE == SIMPLE || (l < k && j < n) ? B[ldb * j + l] : (SRC)0)
                        : (MODE == SIMPLE || (l < k && j < n) ? B[ldb * l + j] : (SRC)0);
            }

        __syncthreads();

        for (int l = 0; l < BK; ++l) {
            for (int ii = 0; ii < WMI; ++ii)
                for (int i = 0; i < TM; ++i)
                    Ar[TM * ii + i] =
                        As[BM * l + WM * warpRow + WSUBM * ii + TM * threadRowInWarp + i];
            for (int jj = 0; jj < WNI; ++jj)
                for (int j = 0; j < TN; ++j)
                    Br[TN * jj + j] =
                        Bs[BN * l + WN * warpCol + WSUBN * jj + TN * threadColInWarp + j];
            for (int ii = 0; ii < WMI; ++ii)
                for (int jj = 0; jj < WNI; ++jj)
                    for (int i = 0; i < TM; ++i)
                        for (int j = 0; j < TN; ++j)
                            Cr[(WNI * TN) * (TM * ii + i) + (TN * jj) + j] +=
                                Ar[TM * ii + i] * Br[TN * jj + j];
        }

        __syncthreads();
    }

    for (int wSubRowIdx = 0; wSubRowIdx < WMI; ++wSubRowIdx)
        for (int wSubColIdx = 0; wSubColIdx < WNI; ++wSubColIdx)
            for (int resIdxM = 0; resIdxM < TM; resIdxM += 1)
                for (int resIdxN = 0; resIdxN < TN; resIdxN += 1) {
                    int row = (BM * blockIdx.y + WM * warpRow) + (WSUBM * wSubRowIdx) +
                              (threadRowInWarp * TM + resIdxM);
                    int col = (BN * blockIdx.x + WN * warpCol) + (WSUBN * wSubColIdx) +
                              (threadColInWarp * TN + resIdxN);
                    if (MODE == SIMPLE) {
                        if (row < m)
                            C[ldc * row + col] = Cr[(WNI * TN) * (TM * wSubRowIdx + resIdxM) +
                                                    TN * wSubColIdx + resIdxN];
                    } else {
                        if (row < m && col < n)
                            C[ldc * row + col] =
                                alpha * Cr[(WNI * TN) * (TM * wSubRowIdx + resIdxM) +
                                           TN * wSubColIdx + resIdxN] +
                                beta * (WORD)C[ldc * row + col];
                    }
                }
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// tinyBLAS canonical cuBLAS-like interface

struct tinyblasContext {
    hipStream_t stream;
};

/**
 * Creates new tinyBLAS handle.
 *
 * Before calling tinyBLAS GEMM functions a handle must first be
 * created, using this function. It should be freed later, using
 * tinyblasDestroy(). After a handle is created the caller needs
 * tinyblasSetStream() to specify the CUDA stream.
 *
 * @param out_handle receives pointer to newly created handle
 * @return TINYBLAS_STATUS_SUCCESS on success otherwise error
 */
tinyblasStatus_t tinyblasCreate(tinyblasHandle_t *out_handle) {
    tinyblasHandle_t handle;
    if ((handle = (tinyblasHandle_t)malloc(sizeof(struct tinyblasContext)))) {
        *out_handle = handle;
        return TINYBLAS_STATUS_SUCCESS;
    } else {
        return TINYBLAS_STATUS_ALLOC_FAILED;
    }
}

/**
 * Destroys tinyBLAS handle.
 *
 * @param handle is pointer to handle created by tinyblasCreate()
 * @return TINYBLAS_STATUS_SUCCESS on success otherwise error
 */
tinyblasStatus_t tinyblasDestroy(tinyblasHandle_t handle) {
    free(handle);
    return TINYBLAS_STATUS_SUCCESS;
}

/**
 * Associates CUDA handle with tinyBLAS handle.
 *
 * The provided stream will be used when tinyBLAS launches kernels.
 *
 * @param handle is pointer to handle created by tinyblasCreate()
 * @param stream is pointer to stream created by hipStreamCreate()
 * @return TINYBLAS_STATUS_SUCCESS on success otherwise error
 */
tinyblasStatus_t tinyblasSetStream(tinyblasHandle_t handle, void *stream) {
    handle->stream = (hipStream_t)stream;
    return TINYBLAS_STATUS_SUCCESS;
}

/**
 * Gets CUDA stream associated with tinyBLAS handle.
 *
 * @param handle is pointer to handle created by tinyblasCreate()
 * @param out_stream receives pointer to any hipStream_t object
 * @return TINYBLAS_STATUS_SUCCESS on success otherwise error
 */
tinyblasStatus_t tinyblasGetStream(tinyblasHandle_t handle, void **out_stream) {
    *out_stream = handle->stream;
    return TINYBLAS_STATUS_SUCCESS;
}

/**
 * Returns string describing tinyBLAS status code.
 */
const char *tinyblasGetStatusString(tinyblasStatus_t err) {
    switch (err) {
    case TINYBLAS_STATUS_SUCCESS:
        return "Success";
    case TINYBLAS_STATUS_ALLOC_FAILED:
        return "Alloc failed";
    case TINYBLAS_STATUS_INVALID_VALUE:
        return "Invalid value";
    case TINYBLAS_STATUS_NOT_SUPPORTED:
        return "Not supported";
    case TINYBLAS_STATUS_EXECUTION_FAILED:
        return "Execution failed";
    case TINYBLAS_STATUS_DIMENSION_OVERFLOW:
        return "Dimension overflow";
    default:
        return "Unknown error";
    }
}

/**
 * Performs single-precision general matrix multiplication.
 *
 * This is a column major GEMM subroutine for computing C = α*A*B + β*C.
 *
 * @param handle was created by tinyblasCreate()
 * @param transa if `A` should be transposed
 * @param transb if `B` should be transposed
 * @param m is rows in `A` and `C`
 * @param n is cols in `B` and `C`
 * @param k is cols in `A` and rows in `B`
 * @param alpha points to scalar that's multiplied against input
 * @param A is input array of first matrix
 * @param lda is row stride of `A`
 * @param B is input array of second matrix
 * @param ldb is row stride of `B`
 * @param beta points to scalar that's multiplied against existing output
 * @param C is input/output array of output matrix
 * @param ldc is row stride of `C`
 */
tinyblasStatus_t tinyblasSgemm(tinyblasHandle_t handle, tinyblasOperation_t transa,
                               tinyblasOperation_t transb, int m, int n, int k, const float *alpha,
                               const float *A, int lda, const float *B, int ldb, const float *beta,
                               float *C, int ldc) {
    return tinyblasGemmEx(handle, transa, transb, m, n, k, alpha, A, TINYBLAS_R_32F, lda, B,
                          TINYBLAS_R_32F, ldb, beta, C, TINYBLAS_R_32F, ldc, TINYBLAS_COMPUTE_32F,
                          TINYBLAS_GEMM_DEFAULT);
}

template <enum Mode MODE, int BM, int BN, int BK, int WM, int WN, int WNI, int TM, int TN, int TT,
          typename WORD, typename SRC, typename DST>
static __global__ void __launch_bounds__(TT) tinyblasGE_entry(tinyblasOperation_t aT, //
                                                              tinyblasOperation_t bT, //
                                                              int m, int n, int k, WORD alpha, //
                                                              const SRC *A, int lda, //
                                                              const SRC *B, int ldb, //
                                                              WORD beta, DST *C, int ldc) {
    matmul_warp2d<MODE, BM, BN, BK, WM, WN, WNI, TM, TN, TT>(aT, bT, m, n, k, alpha, A, lda, B, ldb,
                                                             beta, C, ldc);
}

template <typename WORD, typename SRC, typename DST>
static tinyblasStatus_t tinyblasGE_launcher(tinyblasHandle_t handle, tinyblasOperation_t aT,
                                            tinyblasOperation_t bT, int m, int n, int k, WORD alpha,
                                            const SRC *A, int lda, const SRC *B, int ldb, WORD beta,
                                            DST *C, int ldc) {
    const int TT = 256, BM = 128, BN = 64, BK = 64, WM = 128, WN = 8, WNI = 1, TM = 8, TN = 4;
    dim3 maxblocks(CEIL_DIV(m, BM), CEIL_DIV(n, BN));
    if (!aT && bT && !(n % BN) && !(k % BK) && alpha == (WORD)1 && beta == (WORD)0) {
        dim3 blocks(CEIL_DIV(n, BN), CEIL_DIV(m, BM));
        tinyblasGE_entry<SIMPLE, BM, BN, BK, WM, WN, WNI, TM, TN, TT>
            <<<blocks, TT, 0, handle->stream>>>(aT, bT, m, n, k, alpha, A, lda, B, ldb, beta, C,
                                                ldc);
    } else {
        dim3 blocks(CEIL_DIV(n, BN), CEIL_DIV(m, BM));
        tinyblasGE_entry<GENERAL, BM, BN, BK, WM, WN, WNI, TM, TN, TT>
            <<<blocks, TT, 0, handle->stream>>>(aT, bT, m, n, k, alpha, A, lda, B, ldb, beta, C,
                                                ldc);
    }
    if (hipGetLastError() != hipSuccess)
        return TINYBLAS_STATUS_EXECUTION_FAILED;
    return TINYBLAS_STATUS_SUCCESS;
}

template <typename WORD, typename SRC, typename DST>
static tinyblasStatus_t tinyblasGE_launch(tinyblasHandle_t handle, tinyblasOperation_t aT,
                                          tinyblasOperation_t bT, int m, int n, int k, WORD alpha,
                                          const SRC *A, int lda, const SRC *B, int ldb, WORD beta,
                                          DST *C, int ldc) {
    return tinyblasGE_launcher(handle, bT, aT, n, m, k, alpha, B, ldb, A, lda, beta, C, ldc);
}

/**
 * Performs extended general matrix multiplication.
 *
 * This is a column major GEMM subroutine for computing C = α*A*B + β*C.
 *
 * @param handle was created by tinyblasCreate()
 * @param transa if `A` should be transposed
 * @param transb if `B` should be transposed
 * @param m is rows in `A` and `C`
 * @param n is cols in `B` and `C`
 * @param k is cols in `A` and rows in `B`
 * @param alpha points to scalar that's multiplied against input
 * @param A is input array of first matrix
 * @param Atype is data type of `C`
 * @param lda is row stride of `A`
 * @param B is input array of second matrix
 * @param Btype is data type of `C`
 * @param ldb is row stride of `B`
 * @param beta points to scalar that's multiplied against existing output
 * @param C is input/output array of output matrix
 * @param Ctype is data type of `C`
 * @param ldc is row stride of `C`
 * @param computeType is data type of `alpha`, `beta`, and dot product
 * @param algo specifies algorithm to use
 */
tinyblasStatus_t tinyblasGemmEx(tinyblasHandle_t handle, //
                                tinyblasOperation_t transa, //
                                tinyblasOperation_t transb, //
                                int m, int n, int k, //
                                const void *alpha, //
                                const void *A, tinyblasDataType_t Atype, int lda, //
                                const void *B, tinyblasDataType_t Btype, int ldb, //
                                const void *beta, //
                                void *C, tinyblasDataType_t Ctype, int ldc, //
                                tinyblasComputeType_t computeType, //
                                tinyblasGemmAlgo_t algo) {

    if (m < 0 || n < 0 || k < 0)
        return TINYBLAS_STATUS_INVALID_VALUE;
    if (lda < std::max(1, transa ? k : m))
        return TINYBLAS_STATUS_INVALID_VALUE;
    if (ldb < std::max(1, transb ? n : k))
        return TINYBLAS_STATUS_INVALID_VALUE;
    if (ldc < std::max(1, m))
        return TINYBLAS_STATUS_INVALID_VALUE;
    if (1ll * lda * ((transa ? k : m) - 1) + ((transa ? m : k) - 1) > INT_MAX)
        return TINYBLAS_STATUS_DIMENSION_OVERFLOW;
    if (1ll * ldb * ((transb ? n : k) - 1) + ((transb ? k : n) - 1) > INT_MAX)
        return TINYBLAS_STATUS_DIMENSION_OVERFLOW;
    if (1ll * ldc * (n - 1) + (m - 1) > INT_MAX)
        return TINYBLAS_STATUS_DIMENSION_OVERFLOW;
    if (algo != TINYBLAS_GEMM_DEFAULT)
        return TINYBLAS_STATUS_INVALID_VALUE;
    if (Atype != Btype)
        return TINYBLAS_STATUS_NOT_SUPPORTED;

    switch (Atype) {
    case TINYBLAS_R_16F:
        switch (Ctype) {
        case TINYBLAS_R_16F:
            switch (computeType) {
            case TINYBLAS_COMPUTE_16F:
                return tinyblasGE_launch(
                    handle, transa, transb, m, n, k, (float)*(const half *)alpha, (const half *)A,
                    lda, (const half *)B, ldb, (float)*(const half *)beta, (half *)C, ldc);
            case TINYBLAS_COMPUTE_32F:
                return tinyblasGE_launch(handle, transa, transb, m, n, k, *(const float *)alpha,
                                         (const half *)A, lda, (const half *)B, ldb,
                                         *(const float *)beta, (half *)C, ldc);
            default:
                return TINYBLAS_STATUS_INVALID_VALUE;
            }
        case TINYBLAS_R_32F:
            switch (computeType) {
            case TINYBLAS_COMPUTE_16F:
                return TINYBLAS_STATUS_NOT_SUPPORTED;
            case TINYBLAS_COMPUTE_32F:
                return tinyblasGE_launch(handle, transa, transb, m, n, k, *(const float *)alpha,
                                         (const half *)A, lda, (const half *)B, ldb,
                                         *(const float *)beta, (float *)C, ldc);
            default:
                return TINYBLAS_STATUS_INVALID_VALUE;
            }
        default:
            return TINYBLAS_STATUS_INVALID_VALUE;
        }
    case TINYBLAS_R_32F:
        switch (Ctype) {
        case TINYBLAS_R_16F:
            return TINYBLAS_STATUS_NOT_SUPPORTED;
        case TINYBLAS_R_32F:
            switch (computeType) {
            case TINYBLAS_COMPUTE_16F:
                return TINYBLAS_STATUS_NOT_SUPPORTED;
            case TINYBLAS_COMPUTE_32F:
                return tinyblasGE_launch(handle, transa, transb, m, n, k, *(const float *)alpha,
                                         (const float *)A, lda, (const float *)B, ldb,
                                         *(const float *)beta, (float *)C, ldc);
            default:
                return TINYBLAS_STATUS_INVALID_VALUE;
            }
        default:
            return TINYBLAS_STATUS_INVALID_VALUE;
        }
    default:
        return TINYBLAS_STATUS_INVALID_VALUE;
    }
}

template <int BM, int BN, int TM, int TN, typename WORD, typename SRC, typename DST>
static __global__ void KERNEL tinyblasGBE_entry(tinyblasOperation_t transa,
                                                tinyblasOperation_t transb, int m, int n, int k,
                                                WORD alpha, const SRC *const Aarray[], int lda,
                                                const SRC *const Barray[], int ldb, WORD beta,
                                                DST *const Carray[], int ldc, int batchCount) {
    for (int z = blockIdx.z; z < batchCount; z += gridDim.z)
        matmul_block2d<BM, BN, TM, TN>(transa, transb, m, n, k, alpha, Aarray[z], lda, Barray[z],
                                       ldb, beta, Carray[z], ldc);
}

template <typename WORD, typename SRC, typename DST>
static tinyblasStatus_t tinyblasGBE_launch(tinyblasHandle_t handle, tinyblasOperation_t transa,
                                           tinyblasOperation_t transb, int m, int n, int k,
                                           WORD alpha, const SRC *const *Aarray, int lda,
                                           const SRC *const *Barray, int ldb, WORD beta,
                                           DST *const *Carray, int ldc, int batchCount) {
    constexpr int BC = 32;
    constexpr int BM = 32;
    constexpr int BN = 32;
    constexpr int TM = 2;
    constexpr int TN = 8;
    dim3 maxblocks(CEIL_DIV(m, BM), CEIL_DIV(n, BN), BC);
    tinyblasGBE_entry<BM, BN, TM, TN><<<maxblocks, THREAD_COUNT, 0, handle->stream>>>(
        transa, transb, m, n, k, alpha, Aarray, lda, Barray, ldb, beta, Carray, ldc, batchCount);
    if (hipGetLastError() != hipSuccess)
        return TINYBLAS_STATUS_EXECUTION_FAILED;
    return TINYBLAS_STATUS_SUCCESS;
}

tinyblasStatus_t tinyblasGemmBatchedEx(tinyblasHandle_t handle, tinyblasOperation_t transa,
                                       tinyblasOperation_t transb, int m, int n, int k,
                                       const void *alpha, const void *const Aarray[],
                                       tinyblasDataType_t Atype, int lda,
                                       const void *const Barray[], tinyblasDataType_t Btype,
                                       int ldb, const void *beta, void *const Carray[],
                                       tinyblasDataType_t Ctype, int ldc, int batchCount,
                                       tinyblasComputeType_t computeType, tinyblasGemmAlgo_t algo) {

    if (m < 0 || n < 0 || k < 0)
        return TINYBLAS_STATUS_INVALID_VALUE;
    if (lda < std::max(1, transa ? k : m))
        return TINYBLAS_STATUS_INVALID_VALUE;
    if (ldb < std::max(1, transb ? n : k))
        return TINYBLAS_STATUS_INVALID_VALUE;
    if (ldc < std::max(1, m))
        return TINYBLAS_STATUS_INVALID_VALUE;
    if (1ll * lda * ((transa ? k : m) - 1) + ((transa ? m : k) - 1) > INT_MAX)
        return TINYBLAS_STATUS_DIMENSION_OVERFLOW;
    if (1ll * ldb * ((transb ? n : k) - 1) + ((transb ? k : n) - 1) > INT_MAX)
        return TINYBLAS_STATUS_DIMENSION_OVERFLOW;
    if (1ll * ldc * (n - 1) + (m - 1) > INT_MAX)
        return TINYBLAS_STATUS_DIMENSION_OVERFLOW;
    if (algo != TINYBLAS_GEMM_DEFAULT)
        return TINYBLAS_STATUS_INVALID_VALUE;
    if (Atype != Btype)
        return TINYBLAS_STATUS_NOT_SUPPORTED;

    switch (Atype) {
    case TINYBLAS_R_16F:
        switch (Ctype) {
        case TINYBLAS_R_16F:
            switch (computeType) {
            case TINYBLAS_COMPUTE_16F:
                return tinyblasGBE_launch(
                    handle, transa, transb, m, n, k, (float)*(const half *)alpha,
                    (const half *const *)Aarray, lda, (const half *const *)Barray, ldb,
                    (float)*(const half *)beta, (half *const *)Carray, ldc, batchCount);
            case TINYBLAS_COMPUTE_32F:
                return tinyblasGBE_launch(handle, transa, transb, m, n, k, *(const float *)alpha,
                                          (const half *const *)Aarray, lda,
                                          (const half *const *)Barray, ldb, *(const float *)beta,
                                          (half *const *)Carray, ldc, batchCount);
            default:
                return TINYBLAS_STATUS_INVALID_VALUE;
            }
        case TINYBLAS_R_32F:
            switch (computeType) {
            case TINYBLAS_COMPUTE_16F:
                return TINYBLAS_STATUS_NOT_SUPPORTED;
            case TINYBLAS_COMPUTE_32F:
                return tinyblasGBE_launch(handle, transa, transb, m, n, k, *(const float *)alpha,
                                          (const half *const *)Aarray, lda,
                                          (const half *const *)Barray, ldb, *(const float *)beta,
                                          (float *const *)Carray, ldc, batchCount);
            default:
                return TINYBLAS_STATUS_INVALID_VALUE;
            }
        default:
            return TINYBLAS_STATUS_INVALID_VALUE;
        }
    case TINYBLAS_R_32F:
        switch (Ctype) {
        case TINYBLAS_R_16F:
            return TINYBLAS_STATUS_NOT_SUPPORTED;
        case TINYBLAS_R_32F:
            switch (computeType) {
            case TINYBLAS_COMPUTE_16F:
                return TINYBLAS_STATUS_NOT_SUPPORTED;
            case TINYBLAS_COMPUTE_32F:
                return tinyblasGBE_launch(handle, transa, transb, m, n, k, *(const float *)alpha,
                                          (const float *const *)Aarray, lda,
                                          (const float *const *)Barray, ldb, *(const float *)beta,
                                          (float *const *)Carray, ldc, batchCount);
            default:
                return TINYBLAS_STATUS_INVALID_VALUE;
            }
        default:
            return TINYBLAS_STATUS_INVALID_VALUE;
        }
    default:
        return TINYBLAS_STATUS_INVALID_VALUE;
    }
}

template <int BM, int BN, int TM, int TN, typename SRC, typename DST, typename WORD>
static __global__ void KERNEL tinyblasGSBE_entry(tinyblasOperation_t transa,
                                                 tinyblasOperation_t transb, int m, int n, int k,
                                                 WORD alpha, const SRC *A, int lda,
                                                 long long strideA, const SRC *B, int ldb,
                                                 long long strideB, WORD beta, DST *C, int ldc,
                                                 long long strideC, int batchCount) {
    matmul_block2d<BM, BN, TM, TN>(transa, transb, m, n, k, alpha, A + blockIdx.z * strideA, lda,
                                   B + blockIdx.z * strideB, ldb, beta, C + blockIdx.z * strideC,
                                   ldc);
}

template <typename WORD, typename SRC, typename DST>
static tinyblasStatus_t tinyblasGSBE_launch(tinyblasHandle_t handle, tinyblasOperation_t transa,
                                            tinyblasOperation_t transb, int m, int n, int k,
                                            WORD alpha, const SRC *A, int lda, long long strideA,
                                            const SRC *B, int ldb, long long strideB, WORD beta,
                                            DST *C, int ldc, long long strideC, int batchCount) {
    constexpr int BC = 32;
    constexpr int BM = 32;
    constexpr int BN = 32;
    constexpr int TM = 2;
    constexpr int TN = 8;
    dim3 maxblocks(CEIL_DIV(m, BM), CEIL_DIV(n, BN), BC);
    tinyblasGSBE_entry<BM, BN, TM, TN><<<maxblocks, THREAD_COUNT, 0, handle->stream>>>(
        transa, transb, m, n, k, alpha, A, lda, strideA, B, ldb, strideB, beta, C, ldc, strideC,
        batchCount);
    if (hipGetLastError() != hipSuccess)
        return TINYBLAS_STATUS_EXECUTION_FAILED;
    return TINYBLAS_STATUS_SUCCESS;
}

tinyblasStatus_t tinyblasGemmStridedBatchedEx(tinyblasHandle_t handle, //
                                              tinyblasOperation_t transa, //
                                              tinyblasOperation_t transb, //
                                              int m, int n, int k, //
                                              const void *alpha, //
                                              const void *A, tinyblasDataType_t Atype, int lda,
                                              long long strideA, //
                                              const void *B, tinyblasDataType_t Btype, int ldb,
                                              long long strideB, //
                                              const void *beta, //
                                              void *C, tinyblasDataType_t Ctype, int ldc,
                                              long long strideC, //
                                              int batchCount, //
                                              tinyblasComputeType_t computeType, //
                                              tinyblasGemmAlgo_t algo) {

    if (m < 0 || n < 0 || k < 0)
        return TINYBLAS_STATUS_INVALID_VALUE;
    if (lda < std::max(1, transa ? k : m))
        return TINYBLAS_STATUS_INVALID_VALUE;
    if (ldb < std::max(1, transb ? n : k))
        return TINYBLAS_STATUS_INVALID_VALUE;
    if (ldc < std::max(1, m))
        return TINYBLAS_STATUS_INVALID_VALUE;
    if (1ll * lda * ((transa ? k : m) - 1) + ((transa ? m : k) - 1) > INT_MAX)
        return TINYBLAS_STATUS_DIMENSION_OVERFLOW;
    if (1ll * ldb * ((transb ? n : k) - 1) + ((transb ? k : n) - 1) > INT_MAX)
        return TINYBLAS_STATUS_DIMENSION_OVERFLOW;
    if (1ll * ldc * (n - 1) + (m - 1) > INT_MAX)
        return TINYBLAS_STATUS_DIMENSION_OVERFLOW;
    if (algo != TINYBLAS_GEMM_DEFAULT)
        return TINYBLAS_STATUS_INVALID_VALUE;
    if (Atype != Btype)
        return TINYBLAS_STATUS_NOT_SUPPORTED;

    switch (Atype) {
    case TINYBLAS_R_16F:
        switch (Ctype) {
        case TINYBLAS_R_16F:
            switch (computeType) {
            case TINYBLAS_COMPUTE_16F:
                return tinyblasGSBE_launch(
                    handle, transa, transb, m, n, k, (float)*(const half *)alpha, (const half *)A,
                    lda, strideA, (const half *)B, ldb, strideB, (float)*(const half *)beta,
                    (half *)C, ldc, strideC, batchCount);
            case TINYBLAS_COMPUTE_32F:
                return tinyblasGSBE_launch(handle, transa, transb, m, n, k, *(const float *)alpha,
                                           (const half *)A, lda, strideA, (const half *)B, ldb,
                                           strideB, *(const float *)beta, (half *)C, ldc, strideC,
                                           batchCount);
            default:
                return TINYBLAS_STATUS_INVALID_VALUE;
            }
        case TINYBLAS_R_32F:
            switch (computeType) {
            case TINYBLAS_COMPUTE_16F:
                return TINYBLAS_STATUS_NOT_SUPPORTED;
            case TINYBLAS_COMPUTE_32F:
                return tinyblasGSBE_launch(handle, transa, transb, m, n, k, *(const float *)alpha,
                                           (const half *)A, lda, strideA, (const half *)B, ldb,
                                           strideB, *(const float *)beta, (float *)C, ldc, strideC,
                                           batchCount);
            default:
                return TINYBLAS_STATUS_INVALID_VALUE;
            }
        default:
            return TINYBLAS_STATUS_INVALID_VALUE;
        }
    case TINYBLAS_R_32F:
        switch (Ctype) {
        case TINYBLAS_R_16F:
            return TINYBLAS_STATUS_NOT_SUPPORTED;
        case TINYBLAS_R_32F:
            switch (computeType) {
            case TINYBLAS_COMPUTE_16F:
                return TINYBLAS_STATUS_NOT_SUPPORTED;
            case TINYBLAS_COMPUTE_32F:
                return tinyblasGSBE_launch(handle, transa, transb, m, n, k, *(const float *)alpha,
                                           (const float *)A, lda, strideA, (const float *)B, ldb,
                                           strideB, *(const float *)beta, (float *)C, ldc, strideC,
                                           batchCount);
            default:
                return TINYBLAS_STATUS_INVALID_VALUE;
            }
        default:
            return TINYBLAS_STATUS_INVALID_VALUE;
        }
    default:
        return TINYBLAS_STATUS_INVALID_VALUE;
    }
}
