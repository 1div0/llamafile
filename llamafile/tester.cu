#include "hip/hip_runtime.h"
// -*- mode:c++;indent-tabs-mode:nil;c-basic-offset:4;coding:utf-8 -*-
// vi: set et ft=c++ ts=4 sts=4 sw=4 fenc=utf-8 :vi
//
// Copyright 2024 Mozilla Foundation
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "tester.h"

#include <atomic>
#include <ctime>
#include <unistd.h>

thread_local const char *is_self_testing;
const size_t kPageSize = std::max(sysconf(_SC_PAGESIZE), 4096l);

long long micros(void) {
    struct timespec ts;
    clock_gettime(CLOCK_REALTIME, &ts);
    return ts.tv_sec * 1000000 + (ts.tv_nsec + 999) / 1000;
}

int rand32(void) {
    static unsigned long long lcg = 1;
    lcg *= 6364136223846793005;
    lcg += 1442695040888963407;
    return lcg >> 32;
}

float float01(unsigned x) { // (0,1)
    return 1.f / 8388608 * ((x >> 9) + .5f);
}

float numba(void) { // (-1,1)
    return float01(rand32()) * 2 - 1;
}

void *cudaMallocManagedOrDie(size_t n) {
    void *p;
    CUDA_OR_DIE(hipMallocManaged(&p, n));
    return p;
}

void cudaFreeOrDie(void *p) {
    CUDA_OR_DIE(hipFree(p));
}

[[noreturn]] void cuda_die(const char *stmt, const char *func, const char *file, int line,
                           const char *msg) {
    int id = -1;
    (void)hipGetDevice(&id);
    fprintf(stderr, "CUDA error: %s\n", msg);
    fprintf(stderr, "  current device: %d, in function %s at %s:%d\n", id, func, file, line);
    fprintf(stderr, "  %s\n", stmt);
    (void)hipDeviceReset();
    _Exit(1);
}

void test_matmul(std::function<void(int m, int n, int k, int l, float α, float β)> f) {
    static const int kDims[] = {1, 2, 23, 77, 15, 2048, 512, 127, 129, 128, 16};
    static const float kAlphas[] = {1, .1};
    static const float kBetas[] = {0, .1};
    static const int kLeads[] = {0, 1};
    std::atomic_llong t = ATOMIC_VAR_INIT(0);
    for (int mi = 0; mi < ARRAYLEN(kDims); ++mi)
        for (int ni = 0; ni < ARRAYLEN(kDims); ++ni)
            for (int li = 0; li < ARRAYLEN(kLeads); ++li)
                for (int ki = 0; ki < ARRAYLEN(kDims); ++ki) {
                    int m = kDims[mi];
                    int n = kDims[ARRAYLEN(kDims) - 1 - ni];
                    int k = kDims[ki];
                    int l = kLeads[li];
                    char name[256];
                    sprintf(name, "testing %4d %4d %4d ld+%d", m, n, k, l);
                    if (t++ % 7 == 0)
                        fprintf(stderr, "%s\r", name);
                    is_self_testing = name;
                    f(m, n, k, l, 1, 0);
                    is_self_testing = 0;
                }
    for (int ai = 0; ai < ARRAYLEN(kAlphas); ++ai)
        for (int bi = 0; bi < ARRAYLEN(kBetas); ++bi) {
            float α = kAlphas[ai];
            float β = kBetas[bi];
            int m = 128;
            int n = 128;
            int k = 128;
            char name[256];
            sprintf(name, "testing %4d %4d %4d α=%g β=%g", m, n, k, α, β);
            if (t++ % 7 == 0)
                fprintf(stderr, "%s\r", name);
            is_self_testing = name;
            f(m, n, k, 0, α, β);
            is_self_testing = 0;
        }
}

static int cuda_tester_init() {
    CUDA_OR_DIE(hipSetDevice(0));
#ifdef __HIP_PLATFORM_AMD__
    rocblas_initialize();
#endif
    CUDA_OR_DIE(hipDeviceSynchronize());
    return 0;
}

const int cuda_tester_ = cuda_tester_init();
